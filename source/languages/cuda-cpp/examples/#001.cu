
#include <hip/hip_runtime.h>
// Kernel definition
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
 {
     int i = blockDim.x * blockIdx.x + threadIdx.x;

     if (i < numElements)
     {
         C[i] = A[i] + B[i];
     }
 }

 int main(void)
 {
     // Initialize some stuff...
     float *d_A = NULL;
     float *d_B = NULL;
     float *d_C = NULL;

     // Do some stuff...

     // Launch the kernel
     int numElements = 50000;
     int threadsPerBlock = 256;
     int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;

     vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

     // Clean up...

     return 0;
 }
